
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void matrixMultiply(int *A, int *B, int *C, int A_rows, int A_cols, int B_cols) {
    // Multiply A (A_rows x A_cols) with B (A_cols x B_cols) and store result in C (A_rows x B_cols)
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < B_cols; j++) {
            C[i * B_cols + j] = 0;
            for (int k = 0; k < A_cols; k++) {
                C[i * B_cols + j] += A[i * A_cols + k] * B[k * B_cols + j];
            }
        }
    }
}

void printMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    // Define the dimensions of matrix A (2x3) and matrix B (3x2)
    int A_rows = 2, A_cols = 3;
    int B_rows = 3, B_cols = 2;

    // Allocate memory for matrices A, B, and C
    int *A = (int *)malloc(A_rows * A_cols * sizeof(int));
    int *B = (int *)malloc(B_rows * B_cols * sizeof(int));
    int *C = (int *)malloc(A_rows * B_cols * sizeof(int));

    // Initialize matrix A (2x3)
    A[0] = 1; A[1] = 2; A[2] = 3;
    A[3] = 4; A[4] = 5; A[5] = 6;

    // Initialize matrix B (3x2)
    B[0] = 7; B[1] = 8;
    B[2] = 9; B[3] = 10;
    B[4] = 11; B[5] = 12;

    // Call matrix multiplication function
    matrixMultiply(A, B, C, A_rows, A_cols, B_cols);

    // Output the result matrix C
    printf("Result matrix C (A * B):\n");
    printMatrix(C, A_rows, B_cols);

    // Free dynamically allocated memory
    free(A);
    free(B);
    free(C);

    return 0;
}