
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void matrixAdd(int *A, int *B, int *C, int rows, int cols) {
    // Perform element-wise matrix addition
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            C[i * cols + j] = A[i * cols + j] + B[i * cols + j];
        }
    }
}

void printMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    // Define the dimensions of matrix A and matrix B (both 3x3 for example)
    int rows = 3, cols = 3;

    // Allocate memory for matrices A, B, and C (the result matrix)
    int *A = (int *)malloc(rows * cols * sizeof(int));
    int *B = (int *)malloc(rows * cols * sizeof(int));
    int *C = (int *)malloc(rows * cols * sizeof(int));

    // Initialize matrix A (3x3)
    A[0] = 1; A[1] = 2; A[2] = 3;
    A[3] = 4; A[4] = 5; A[5] = 6;
    A[6] = 7; A[7] = 8; A[8] = 9;

    // Initialize matrix B (3x3)
    B[0] = 9; B[1] = 8; B[2] = 7;
    B[3] = 6; B[4] = 5; B[5] = 4;
    B[6] = 3; B[7] = 2; B[8] = 1;

    // Perform matrix addition
    matrixAdd(A, B, C, rows, cols);

    // Print the result matrix C
    printf("Result matrix C (A + B):\n");
    printMatrix(C, rows, cols);

    // Free the allocated memory
    free(A);
    free(B);
    free(C);

    return 0;
}
